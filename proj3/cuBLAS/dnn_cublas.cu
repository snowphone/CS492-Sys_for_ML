#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stddef.h>

extern "C"

void conv2D(int PW, int PH, int KW, int KH, int IC, int OC, int SW, int SH, int OW, int OH, float *I, float *W, float *O)
{
	// cuBLAS implementation of 2D convolution
	// cuBLAS's Sgemm is executed based on that the matrix are ordered in Column Major.
	// Store matA and matB in column major order and caculate matrix multiplication and store the result to matC.
	// After done convolution, store matC to Output matrix in row major order.

	// Since unexpected calculation error occurs when use transpose approach to handle RowMajor <-> ColumnMajor,
	// Just reuse openBLAS code that properly working but different major.
	
	int o_size = OW * OH, k_size = KW * KH;
	int r_idx, c_idx, b_idx, w_idx;
	float *matA, *matB, *matC;

	hipblasHandle_t handle;
	float al = 1.0f, bet = 1.0f;
	float *d_a, *d_b, *d_o;
	
	matC = (float *)malloc((o_size * OC) * sizeof(float));
	for(int i = 0; i < o_size * OC; i++)
		matC[i] = 0;	


	for(int ic = 0; ic < IC; ic++)
	{
		matA = (float *)malloc((o_size * k_size) * sizeof(float));

		for(int ow = 0; ow < OW; ow++)
		{
			for(int oh  = 0; oh < OH; oh++)
			{
				r_idx = ow * OH + oh;
				for(int i = 0; i < KW; i++)
				{
					for(int j = 0; j < KH; j++)
					{
						c_idx = i * KH + j;
						matA[c_idx * o_size + r_idx] = I[((ow * SW * PH + oh * SH) + (i * PH + j)) * IC + ic];
					}
				}
			}
		}

		matB = (float *)malloc((k_size * OC) * sizeof(float));
		
		for(int i = 0; i < KW; i++)
		{
			for(int j = 0; j < KH; j++)
			{		
				b_idx = i * KH + j;
				for(int oc = 0; oc < OC; oc++)
				{
					w_idx = (b_idx * (IC * OC)) + ic * OC + oc;
					matB[oc * k_size + b_idx] = W[w_idx];
				}
			}
		}

		hipMalloc(&d_a, (o_size * k_size) * sizeof(float));
		hipMalloc(&d_b, (k_size * OC) * sizeof(float));
		hipMalloc(&d_o, (o_size * OC) * sizeof(float));

		hipblasSetMatrix(o_size, k_size, sizeof(float), matA, o_size, d_a, o_size);
		hipblasSetMatrix(k_size, OC,  sizeof(float), matB, k_size, d_b, k_size);
		hipblasSetMatrix(o_size, OC, sizeof(float), matC, o_size, d_o, o_size);

		hipblasCreate(&handle);

		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
				o_size, OC, k_size,
				&al,
				d_a, o_size,
				d_b, k_size,
				&bet,
				d_o, o_size);
		
		hipblasGetMatrix(o_size, OC, sizeof(float), d_o, o_size, matC, o_size);
		
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_o);
		hipblasDestroy(handle);
		free(matA);
		free(matB);
	
	}
	
	for(int i = 0; i < o_size; i++)
		for(int j = 0; j < OC; j++)
			O[i * OC + j] = matC[j * o_size + i];

	free(matC);
}

extern "C"

void biasAdd(int size, int OC, float *I, float *B, float *O)
{
	for(int i = 0; i < size; i++)
		for(int oc = 0; oc < OC; oc++)
		{
			O[i * OC + oc] = I[i * OC + oc] + B[oc];
		}


/*
	cublasHandle_t handle;
	float *d_i, *d_b, *d_o; 

	float al = 1.0f;
*//*
	cudaMalloc(&d_i, size * OC * sizeof(float));
	cudaMalloc(&d_b, size * OC * sizeof(float));
	cudaMalloc(&d_o, size * OC * sizeof(float));

	cublasSetVector(size * OC, sizeof(float), I, 1, d_i, 1);
		
	cublasSetVector(OC, sizeof(float), B, 1, d_b, 1);
	cublasSetVector(size * OC, sizeof(float), O, 1, d_o, 1);

	for(int i = 0; i < size; i++)
	{
		cublasCreate(&handle); 
		cublasSaxpy(handle, OC, &al, d_b, 1, d_o + i * OC, 1);
       		cublasSaxpy(handle, OC, &al, d_i + i * OC, 1, d_o + i * OC, 1);
		cublasDestroy(handle);
	}	

	cublasGetVector(size * OC, sizeof(float), d_o, 1, O, 1);

	cudaFree(d_i);
	cudaFree(d_b);
	cudaFree(d_o);
	cublasDestroy(handle);
*/	
	/*	
	for(int i = 0; i < size; i++)
	{
		for(int oc = 0; oc < OC; oc++)
		{
			O[i * OC + oc] = I[i * OC + oc] + B[oc];
		cudaMalloc(&d_i, OC * sizeof(float));
		cudaMalloc(&d_b, OC * sizeof(float));
		cudaMalloc(&d_o, OC * sizeof(float));

		cublasSetVector(OC, sizeof(float), I + i * OC, 1, d_i, 1);
		
		cublasSetVector(OC, sizeof(float), B, 1, d_b, 1);
		cublasSetVector(OC, sizeof(float), O + i * OC, 1, d_o, 1);

		cublasCreate(&handle); 

		cublasSaxpy(handle, OC, &al, d_b, 1, d_o, 1);
        	cublasSaxpy(handle, OC, &al, d_i, 1, d_o, 1);

		cublasGetVector(OC, sizeof(float), d_o, 1, O + i * OC, 1);

		cudaFree(d_i);
		cudaFree(d_b);
		cudaFree(d_o);
		cublasDestroy(handle);
	}*/
}

extern "C"

void maxPool2D(int PW, int PH, int KW, int KH, int OC, int SW, int SH, int OW, int OH, float *I, float *O)
{
        float max;
        int o_idx, s_idx, k_idx;

        for(int oc = 0; oc < OC; oc++)
        {
                for(int ow = 0; ow < OW; ow++)
                {
                        for(int oh  = 0; oh < OH; oh++)
                        {
                                o_idx = (ow * OH + oh) * OC + oc;
                                s_idx = (ow * SW * PH + oh * SH) * OC + oc;
                                max = I[s_idx];
                                for(int i = 0; i < KW; i++)
                                {
                                        if(ow * SW + i >= PW)
                                                break;
                                        for(int j = 0; j < KH; j++)
                                        {
                                                if(oh * SH + j >= PH)
                                                        break;
                                                k_idx = (i * PH + j) * OC;
                                                if(I[s_idx + k_idx] > max)
                                                        max = I[s_idx + k_idx];
                                        }
                                }
                                O[o_idx] = max;
                        }
                }
        }
}

extern "C"

void batchNorm(int size, int OC, float *I, float *mean, float *gamma, float *variance, float epsilon, float *O)
{

	for(int i = 0; i < size; i++)
		for(int oc = 0; oc < OC; oc++)
		{
			O[i * OC + oc] = (I[i * OC + oc] - mean[oc]) * gamma[oc] / sqrt(variance[oc] + epsilon);	
		}
/*
        cublasHandle_t handle;

	float coeff;

	float al = 1.0f, mal = -1.0f; 	

	float *d_i, *d_m, *d_o; 
	
        for(int i = 0; i < size; i++)
        {
		cudaMalloc(&d_i, OC * sizeof(float));
		cudaMalloc(&d_m, OC * sizeof(float));
		cudaMalloc(&d_o, OC * sizeof(float));

		cublasSetVector(OC, sizeof(float), I + i * OC, 1, d_i, 1);
		cublasSetVector(OC, sizeof(float), mean, 1, d_m, 1);
		cublasSetVector(OC, sizeof(float), O + i * OC, 1, d_o, 1);
		
		cublasCreate(&handle);
 
                cublasSaxpy(handle, OC, &mal, d_m, 1, d_o, 1);
                cublasSaxpy(handle, OC, &al, d_i, 1, d_i, 1);

		cublasGetVector(OC, sizeof(float), d_o, 1, O + i * OC, 1);

		cudaFree(d_i);
		cudaFree(d_m);
		cudaFree(d_o);
		cublasDestroy(handle);
        }

        for(int oc = 0; oc < OC; oc++)
        {
                coeff = gamma[oc] / sqrt(variance[oc] + epsilon);
		
		cudaMalloc(&d_o, size * sizeof(float));
		
		cublasSetVector(size, sizeof(float), O + oc, OC, d_o, 1);		

		cublasCreate(&handle);

                cublasSscal(handle, size, &coeff, d_o, 1);

		cublasGetVector(size, sizeof(float), d_o, 1, O + oc, OC);

		cudaFree(d_o);
        	
		cublasDestroy(handle);
	}
*/
}

extern "C"

void leakyReLU(int size, int OC, float *I, float *O)
{
        int idx;

        for(int i = 0; i < size; i++)
                for(int oc = 0; oc < OC; oc++)
                {
                        idx = i * OC + oc;
                        if(I[idx] < 0)
                                I[idx] = I[idx] * 0.1;
                        O[idx] = I[idx];
                }
}

